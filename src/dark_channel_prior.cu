#include "hip/hip_runtime.h"
#include "dark_channel_prior.cuh"

__global__ void calcRGBMinimumGPU(unsigned char* const inputImageArr, unsigned char* RGBMinimumArr,
                                  const int imageWidth, const int imageHeight) {

    int channelNum = 3;
    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIdx >= imageWidth || yIdx >= imageHeight) return;

    unsigned char b = inputImageArr[(yIdx * imageWidth + xIdx) * channelNum];
    unsigned char g = inputImageArr[(yIdx * imageWidth + xIdx) * channelNum + 1];
    unsigned char r = inputImageArr[(yIdx * imageWidth + xIdx) * channelNum + 2];

    unsigned char RGBMinimumValue;
    RGBMinimumValue = (b < g) ? b : g;
    RGBMinimumValue = (RGBMinimumValue < r) ? RGBMinimumValue : r;

    RGBMinimumArr[yIdx * imageWidth + xIdx] = RGBMinimumValue;
}


void executeDarkChannelPriorGPU(unsigned char* const inputImageArr, unsigned char* RGBMinimumArr,
                                const int imageWidth, const int imageHeight) {
    // Block and grid settings
    dim3 block(64, 4);
    dim3 grid((imageWidth + block.x - 1) / block.x, (imageHeight + block.y - 1) / block.y);

    calcRGBMinimumGPU<<<grid, block>>>(inputImageArr, RGBMinimumArr, imageWidth, imageHeight);
}
