#include "hip/hip_runtime.h"
#include "dark_channel_prior.cuh"


__global__ void calcRGBMinimumGPU(unsigned char* const inputImageArr, unsigned char* RGBMinimumArr,
                                  const int imageWidth, const int imageHeight) {

    int channelNum = 3;
    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIdx >= imageWidth || yIdx >= imageHeight) return;

    unsigned char b = inputImageArr[(yIdx * imageWidth + xIdx) * channelNum];
    unsigned char g = inputImageArr[(yIdx * imageWidth + xIdx) * channelNum + 1];
    unsigned char r = inputImageArr[(yIdx * imageWidth + xIdx) * channelNum + 2];

    unsigned char RGBMinimumValue;
    RGBMinimumValue = (b < g) ? b : g;
    RGBMinimumValue = (RGBMinimumValue < r) ? RGBMinimumValue : r;

    RGBMinimumArr[yIdx * imageWidth + xIdx] = RGBMinimumValue;
}


__global__ void minimumFilterGPU(unsigned char* RGBMinimumArr, unsigned char* outputImageArr, const int radius,
                                 const int imageWidth, const int imageHeight) {
    int centerXIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int centerYIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (centerXIdx >= imageWidth || centerYIdx >= imageHeight) return;

    // Define neighbor area
    int yLowerBound = (centerYIdx - radius) > 0 ? (centerYIdx - radius) : 0;
    int yUpperBound = (centerYIdx + radius) < imageHeight ? (centerYIdx + radius) : (imageHeight - 1);
    int xLowerBound = (centerXIdx - radius) > 0 ? (centerXIdx - radius) : 0;
    int xUpperBound = (centerXIdx + radius) < imageWidth ? (centerXIdx + radius) : (imageWidth - 1);

    // Get minimum value in the neighbor area
    unsigned char neighborMinimumValue = 255;
    for(int yIdx = yLowerBound; yIdx <= yUpperBound; yIdx++) {
        for (int xIdx = xLowerBound; xIdx <= xUpperBound; xIdx++) {
            unsigned char RGBMinimumValue = RGBMinimumArr[yIdx * imageWidth + xIdx];
            neighborMinimumValue = (neighborMinimumValue > RGBMinimumValue) ? RGBMinimumValue : neighborMinimumValue;
        }
    }
    outputImageArr[centerYIdx * imageWidth + centerXIdx] = neighborMinimumValue;
}


void executeDarkChannelPriorGPU(unsigned char* const inputImageArr, unsigned char* RGBMinimumArr, unsigned char* outputImageArr,
                                const int radius, const int imageWidth, const int imageHeight) {
    // Block and grid settings
    dim3 block(64, 4);
    dim3 grid((imageWidth + block.x - 1) / block.x, (imageHeight + block.y - 1) / block.y);

    calcRGBMinimumGPU<<<grid, block>>>(inputImageArr, RGBMinimumArr, imageWidth, imageHeight);

    minimumFilterGPU<<<grid, block>>>(RGBMinimumArr, outputImageArr, radius, imageWidth, imageHeight);
}
