#include "hip/hip_runtime.h"
#include "remove_haze.cuh"


__global__ void dehazeGPU(unsigned char* const inputValue, unsigned char* const transmission, unsigned char* dehazedValue,
                          const float minTransmission, float* const atmosphericLight,
                          const int imageWidth, const int imageHeight) {

    int channelNum = 3;
    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int zIdx = threadIdx.z;

    if (xIdx >= imageWidth || yIdx >= imageHeight || zIdx >= channelNum) return;

    float transmissionValue = (float)(unsigned char)transmission[yIdx * imageWidth + xIdx] / 255.0;
    transmissionValue = (transmissionValue > minTransmission) ? transmissionValue: minTransmission;

    float dehazed = ((float)(unsigned char)inputValue[(yIdx * imageWidth + xIdx) * channelNum + zIdx] - atmosphericLight[zIdx]) / transmissionValue + atmosphericLight[zIdx];

    if (dehazed > 255) {
        dehazed = 255;
    } else if (dehazed < 0) {
        dehazed = 0;
    }

    dehazedValue[(yIdx * imageWidth + xIdx) * channelNum + zIdx] = dehazed;
}


void executeDehazeGPU(unsigned char* const inputValue, unsigned char* const transmission, unsigned char* dehazedValue,
                      const float minTransmission, float* const atmosphericLight,
                      const int imageWidth, const int imageHeight) {
    // Block and grid settings
    int channelNum = 3;
    dim3 block(64, 4, channelNum);
    dim3 grid((imageWidth + block.x - 1) / block.x, (imageHeight + block.y - 1) / block.y);

    // Copy atmospheric light from host to device
    float *dAtmosphericLight;
    int atmosphericLightMemSize = sizeof(float) * channelNum;
    hipMalloc((void **)&dAtmosphericLight, atmosphericLightMemSize);
    hipMemcpy(dAtmosphericLight, atmosphericLight, atmosphericLightMemSize, hipMemcpyHostToDevice);

    dehazeGPU<<<grid, block>>>(inputValue, transmission, dehazedValue, minTransmission, dAtmosphericLight, imageWidth, imageHeight);
}
