#include "hip/hip_runtime.h"
#include "remove_haze.cuh"
#include <stdio.h>


__global__ void dehazeGPU(unsigned char* const inputValue, unsigned char* const transmission, unsigned char* dehazedValue,
                          const double minTransmission, const double atmosphericB, const double atmosphericG, const double atmosphericR,
                          const int imageWidth, const int imageHeight) {

    int channelNum = 3;
    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIdx >= imageWidth || yIdx >= imageHeight) return;

    double transmissionValue = transmission[yIdx * imageWidth + xIdx] / 255.0;
    transmissionValue = (transmissionValue > minTransmission) ? transmissionValue: minTransmission;

    double dehazedB = ((double)(unsigned char)inputValue[(yIdx * imageWidth + xIdx) * channelNum + 0] - atmosphericB) / transmissionValue + atmosphericB;
    double dehazedG = ((double)(unsigned char)inputValue[(yIdx * imageWidth + xIdx) * channelNum + 1] - atmosphericG) / transmissionValue + atmosphericG;
    double dehazedR = ((double)(unsigned char)inputValue[(yIdx * imageWidth + xIdx) * channelNum + 2] - atmosphericR) / transmissionValue + atmosphericR;

    if (dehazedB > 255) {
        dehazedB = 255;
    } else if (dehazedB < 0) {
        dehazedB = 0;
    }

    if (dehazedG > 255) {
        dehazedG = 255;
    } else if (dehazedG < 0) {
        dehazedG = 0;
    }

    if (dehazedR > 255) {
        dehazedR = 255;
    } else if (dehazedR < 0) {
        dehazedR = 0;
    }

    dehazedValue[(yIdx * imageWidth + xIdx) * channelNum + 0] = dehazedB;
    dehazedValue[(yIdx * imageWidth + xIdx) * channelNum + 1] = dehazedG;
    dehazedValue[(yIdx * imageWidth + xIdx) * channelNum + 2] = dehazedR;
}


void executeDehazeGPU(unsigned char* const inputValue, unsigned char* const transmission, unsigned char* dehazedValue,
                      const double minTransmission, const double atmosphericB, const double atmosphericG, const double atmosphericR,
                      const int imageWidth, const int imageHeight) {

    dim3 block(64, 4);
    dim3 grid((imageWidth + block.x - 1) / block.x, (imageHeight + block.y - 1) / block.y);

    dehazeGPU<<<grid, block>>>(inputValue, transmission, dehazedValue, minTransmission, atmosphericB, atmosphericG, atmosphericR, imageWidth, imageHeight);
}
