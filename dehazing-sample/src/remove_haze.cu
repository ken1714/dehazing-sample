#include "hip/hip_runtime.h"
#include "remove_haze.cuh"


__global__ void dehazeGPU(unsigned char* const inputValue, unsigned char* const transmission, unsigned char* dehazedValue,
                          const double minTransmission, double* const atmosphericLight,
                          const int imageWidth, const int imageHeight) {

    int channelNum = 3;
    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int zIdx = threadIdx.z;

    if (xIdx >= imageWidth || yIdx >= imageHeight || zIdx >= channelNum) return;

    double transmissionValue = transmission[yIdx * imageWidth + xIdx] / 255.0;
    transmissionValue = (transmissionValue > minTransmission) ? transmissionValue: minTransmission;

    double dehazed = ((double)(unsigned char)inputValue[(yIdx * imageWidth + xIdx) * channelNum + zIdx] - atmosphericLight[zIdx]) / transmissionValue + atmosphericLight[zIdx];

    if (dehazed > 255) {
        dehazed = 255;
    } else if (dehazed < 0) {
        dehazed = 0;
    }

    dehazedValue[(yIdx * imageWidth + xIdx) * channelNum + zIdx] = dehazed;
}


void executeDehazeGPU(unsigned char* const inputValue, unsigned char* const transmission, unsigned char* dehazedValue,
                      const double minTransmission, double* const atmosphericLight,
                      const int imageWidth, const int imageHeight) {
    // Block and grid settings
    int channelNum = 3;
    dim3 block(64, 4, channelNum);
    dim3 grid((imageWidth + block.x - 1) / block.x, (imageHeight + block.y - 1) / block.y);

    // Copy atmospheric light from host to device
    double *dAtmosphericLight;
    int atmosphericLightMemSize = sizeof(double) * channelNum;
    hipMalloc((void **)&dAtmosphericLight, atmosphericLightMemSize);
    hipMemcpy(dAtmosphericLight, atmosphericLight, atmosphericLightMemSize, hipMemcpyHostToDevice);

    dehazeGPU<<<grid, block>>>(inputValue, transmission, dehazedValue, minTransmission, dAtmosphericLight, imageWidth, imageHeight);
}
